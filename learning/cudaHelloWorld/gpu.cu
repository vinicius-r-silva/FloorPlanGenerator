#include "hip/hip_runtime.h"


#include <cstdio>

__global__ void printHelloGPU()
{
	printf("Hello World from the GPU\n");
}

void launchKernel(){	
	printHelloGPU<<<1, 1>>>();
	hipDeviceSynchronize();
}