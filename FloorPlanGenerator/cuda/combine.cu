#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>
#include <stdint.h>
#include <algorithm>
#include <filesystem>
#include <fstream>

#include "helper.cuh"
#include "combine.cuh"
#include "common.cuh"
#include "process.h"
#include "../lib/cvHelper.h"
#include "../lib/globals.h"



// Sorry, had to do it this way to make the reduce the cuda kernel registers usage
// #define check_overlap(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right) ((a_up >= b_up && a_up < b_down && a_left < b_right && a_left >= b_left) || (a_up >= b_up && a_up < b_down && a_right >= b_right && a_left <= b_left) || (a_up >= b_up && a_up < b_down && a_right <= b_right && a_right > b_left) || (a_down >= b_down && a_left < b_right && a_left >= b_left && a_up <= b_up) || (a_down >= b_down && a_up <= b_up && a_right <= b_right && a_right > b_left) || (a_left < b_right && a_left >= b_left && a_down > b_up && a_down <= b_down) || (a_right >= b_right && a_down > b_up && a_down <= b_down && a_left <= b_left) || (b_right >= a_right && b_up >= a_up && b_up < a_down && b_left <= a_left) || (b_right >= a_right && b_down > a_up && b_down <= a_down && b_left <= a_left) || (b_up >= a_up && b_up < a_down && b_left >= a_left && b_left < a_right) || (b_up >= a_up && b_up < a_down && b_right > a_left && b_right <= a_right) || (b_down >= a_down && b_left >= a_left && b_left < a_right && b_up <= a_up) || (b_down >= a_down && b_right > a_left && b_right <= a_right && b_up <= a_up) || (b_left >= a_left && b_left < a_right && b_down > a_up && b_down <= a_down) || (a_down > b_up && a_down <= b_down && a_right <= b_right && a_right > b_left) || (b_right > a_left && b_right <= a_right && b_down > a_up && b_down <= a_down))

// __device__
// uint8_t check_overlap(const int a_up, const int a_down, const int a_left, const int a_right, 
// 	const int b_up, const int b_down, const int b_left, const int b_right){
// 	if(((a_down > b_up && a_down <= b_down) ||
// 	(a_up  >= b_up && a_up < b_down)) &&
// 	((a_right > b_left && a_right <= b_right) ||
// 	(a_left  >= b_left && a_left  <  b_right) ||
// 	(a_left  <= b_left && a_right >= b_right))){
// 		return 0;
// 	}

// 	else if(((b_down > a_up && b_down <= a_down) ||
// 	(b_up >= a_up && b_up < a_down)) &&
// 	((b_right > a_left && b_right <= a_right) ||
// 	(b_left  >= a_left && b_left  <  a_right) ||
// 	(b_left  <= a_left && b_right >= a_right))){
// 		return 0;
// 	}

// 	else if(((a_right > b_left && a_right <= b_right) ||
// 	(a_left >= b_left && a_left < b_right)) &&
// 	((a_down > b_up && a_down <= b_down) ||
// 	(a_up  >= b_up && a_up   <  b_down) ||
// 	(a_up  <= b_up && a_down >= b_down))){
// 		return 0;
// 	}

// 	else if(((b_right > a_left && b_right <= a_right) ||
// 	(b_left >= a_left && b_left < a_right)) &&
// 	((b_down > a_up && b_down <= a_down) ||
// 	(b_up  >= a_up && b_up   <  a_down) ||
// 	(b_up  <= a_up && b_down >= a_down))){
// 		return 0;
// 	}

// 	return 1;
// }


__global__ 
void k_createPts(int16_t *d_a, int16_t *d_b, int *d_res, int *d_adj, int *d_conn, const int qtd_a, const int qtd_b, const int a_offset) {
	// Block and thread indexes 	
	// Each blockIdx.x iterates over a fixed number (num_a) of A layouts (blockIdx.y), 
	// that iterates over Nconn connections (blockIdx.z). Each threadIdx.x represents
	// a Layout B design inside the blockIdx.x block 

	//K represents the connection (from 0 to 15, skipping 0, 5, 10 and 15)
	// const int kidx = blockIdx.z; 
	int a_idx = blockIdx.y + a_offset; //layout A index
	int b_idx = blockIdx.x * blockDim.x + threadIdx.x; //layout B index
	const uint64_t res_idx = ((blockIdx.y * qtd_b * gridDim.z) + (b_idx * gridDim.z) + blockIdx.z) * __SIZE_RES;

	// Check bounds
	if(b_idx >= qtd_b || blockIdx.y >= qtd_a){
		return;
	}

	a_idx *= __SIZE_A_DISK;
	b_idx *= __SIZE_B_DISK;

	// Load A into shared memory
	__shared__ int16_t a[__SIZE_A_DISK];
	if(threadIdx.x < __SIZE_A_DISK){
		a[threadIdx.x] = d_a[a_idx + threadIdx.x];
	}
	
	__shared__ int req_adj[__SIZE_ADJ];
	if(threadIdx.x < __SIZE_ADJ){
		req_adj[threadIdx.x] = d_adj[threadIdx.x];
	}

	__shared__ int k;
	if(threadIdx.x == 0){
		k = d_conn[blockIdx.z];
	}

  	__syncthreads();

	// if(a_idx != 0 || b_idx != 0)
	// 	return;

	// if(blockIdx.z > 0)
	// 	return;

	// Load B into local memory
	int16_t b[__SIZE_B_DISK];
	for(int i = 0; i < __SIZE_B_DISK; i++){
		b[i] = d_b[b_idx + i];
	}

	// Extract source and destination connections from k
	int srcConn = (k >> __COMBINE_CONN_SRC_X_SHIFT) & __COMBINE_CONN_BITS;
	int dstConn = (k >> __COMBINE_CONN_DST_X_SHIFT) & __COMBINE_CONN_BITS;



	// printf("1 - %ld - pts:\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n\n", 
	// res_idx,
	// a[0], a[1], a[2], a[3], 
	// a[4], a[5], a[6], a[7], 
	// a[8], a[9], a[10], a[11],
	// b[0], b[1], b[2], b[3], 
	// b[4], b[5], b[6], b[7], 
	// b[8], b[9], b[10], b[11]);

	int src = a[srcConn];
	int dst = b[dstConn];
	const int diffX = src - dst;

	srcConn = (k >> __COMBINE_CONN_SRC_Y_SHIFT) & __COMBINE_CONN_BITS;
	dstConn = (k >> __COMBINE_CONN_DST_Y_SHIFT) & __COMBINE_CONN_BITS;
	src = a[srcConn];
	dst = b[dstConn];
	const int diffY = src - dst;

	// printf("\n\nbx: %d, by: %d, bz: %d, tx: %d, ty: %d, tz: %d\nres: %ld, a_idx: %d, b_idx: %d\nk: %d, src X: %d (%d), src Y: %d (%d), dst X: %d (%d), dst Y: %d (%d)\ndiffX: %d, diffY: %d, \n\n",
	// 		blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z,
	// 		res_idx, blockIdx.y + a_offset, blockIdx.x * blockDim.x + threadIdx.x,
	// 		k,
	// 		(k >> __COMBINE_CONN_SRC_X_SHIFT) & __COMBINE_CONN_BITS, a[(k >> __COMBINE_CONN_SRC_X_SHIFT) & __COMBINE_CONN_BITS],
	// 		(k >> __COMBINE_CONN_SRC_Y_SHIFT) & __COMBINE_CONN_BITS, a[(k >> __COMBINE_CONN_SRC_Y_SHIFT) & __COMBINE_CONN_BITS],
	// 		(k >> __COMBINE_CONN_DST_X_SHIFT) & __COMBINE_CONN_BITS, b[(k >> __COMBINE_CONN_DST_X_SHIFT) & __COMBINE_CONN_BITS],
	// 		(k >> __COMBINE_CONN_DST_Y_SHIFT) & __COMBINE_CONN_BITS, b[(k >> __COMBINE_CONN_DST_Y_SHIFT) & __COMBINE_CONN_BITS],
	// 		diffX, diffY);

	//Move layout B in the X and Y axis by diffX and diffY points
	for(int i = 0; i < __SIZE_B_LAYOUT; i+=2){
		b[i] += diffX;
		b[i + 1] += diffY;
	}

	// Find the bounding box of B
	int minH = 5000, maxH = -5000;
	int minW = 5000, maxW = -5000;
	for(int i = 0; i < __SIZE_B_LAYOUT; i+=4){
		if(b[i + __UP] < minH)
			minH = b[i + __UP];
		if(b[i + __DOWN] > maxH)
			maxH = b[i + __DOWN];
		if(b[i] < minW)
			minW = b[i];
		if(b[i + __RIGHT] > maxW)
			maxW = b[i + __RIGHT];
	}

	//left, up, right, down
	// Find the bounding box of A and check overlaping
	int connections[__COMBINE_N_A + __COMBINE_N_B];
	for(int i = 0; i < __COMBINE_N_A  + __COMBINE_N_B; i++){
		connections[i] = 1 << i;
	}

	for(int i = 0; i < __SIZE_A_LAYOUT; i+=4){
		const int a_left = a[i];
		const int a_up = a[i + __UP];
		const int a_down = a[i + __DOWN];
		const int a_right = a[i + __RIGHT];

		if(a_up < minH)
			minH = a_up;
		if(a_down > maxH)
			maxH = a_down;
		if(a_left < minW)
			minW = a_left;
		if(a_right > maxW)
			maxW = a_right;

		for(int j = 0; j < __SIZE_B_LAYOUT; j+=4){
			const int b_left = b[j];
			const int b_up = b[j + __UP];
			const int b_down = b[j + __DOWN];
			const int b_right = b[j + __RIGHT];

			// if(!check_overlap(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right))
			// 	return;
			
			if(check_adjacency(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right)){
				connections[i/4] |= 1 << (j/4) + __COMBINE_N_A;
				connections[(j/4) + __COMBINE_N_A] |= 1 << (i/4); 
			}
		}
	}

	for(int i = 0; i < __SIZE_A_LAYOUT; i+=4){
		const int a_left = a[i];
		const int a_up = a[i + __UP];
		const int a_down = a[i + __DOWN];
		const int a_right = a[i + __RIGHT];

		for(int j = 0; j < __SIZE_A_LAYOUT; j+=4){
			const int b_left = a[j];
			const int b_up = a[j + __UP];
			const int b_down = a[j + __DOWN];
			const int b_right = a[j + __RIGHT];

			if(check_adjacency(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right)){
				connections[i/4] |= 1 << (j/4);
				connections[j/4] |= 1 << (i/4); 
			}
		}
	}

	for(int i = 0; i < __SIZE_B_LAYOUT; i+=4){
		const int a_left = b[i];
		const int a_up = b[i + __UP];
		const int a_down = b[i + __DOWN];
		const int a_right = b[i + __RIGHT];

		for(int j = 0; j < __SIZE_B_LAYOUT; j+=4){
			const int b_left = b[j];
			const int b_up = b[j + __UP];
			const int b_down = b[j + __DOWN];
			const int b_right = b[j + __RIGHT];

			if(check_adjacency(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right)){
				connections[(i/4) + __COMBINE_N_A] |= 1 << ((j/4) + __COMBINE_N_A);
				connections[(j/4) + __COMBINE_N_A] |= 1 << ((i/4) + __COMBINE_N_A); 
			}
		}
	}

	const int a_rid_idx = a[__SIZE_A_LAYOUT];
	const int b_rid_idx = b[__SIZE_B_LAYOUT];

	int adj[__SIZE_ADJ_TYPES]; //Rid connections from the specific rId
	int adj_count[__SIZE_ADJ_TYPES]; //Idx of each room from the specific rId
	for(int i = 0; i < __SIZE_ADJ_TYPES; i++){
		adj[i] = 0;
		adj_count[i] = 0;
	}

	for(int i = 0; i < __COMBINE_N_A; i++){
		const int rplannyId = (a_rid_idx >> (i * __RID_BITS_SIZE)) & __RID_BITS;
		adj_count[rplannyId] |= 1 << i;
		adj[rplannyId] |= connections[i];
	}
	
	for(int i = 0; i < __COMBINE_N_B; i++){
		const int rplannyId = (b_rid_idx >> (i * __RID_BITS_SIZE)) & __RID_BITS;
		adj_count[rplannyId] |= 1 << (i + __COMBINE_N_A);
		adj[rplannyId] |= connections[i + __COMBINE_N_A];
	}

	for(int i = 0; i < __SIZE_ADJ_TYPES; i++){
		for(int j = 0; j < __SIZE_ADJ_TYPES; j++){
			const int req_adj_idx = i*__SIZE_ADJ_TYPES + j;
			// if(req_adj[req_adj_idx] == REQ_ANY && !(adj[j] & adj_count[i]))
			// 	return;

			// if(req_adj[req_adj_idx] == REQ_ALL && (adj[j] & adj_count[i]) != adj_count[i])
			// 	return;
		}
	}

	for(int i = 0; i < __COMBINE_N_A + __COMBINE_N_B; i++){
		const int conns = connections[i];
		for(int j = i + 1; j < __COMBINE_N_A + __COMBINE_N_B; j++){
			if(connections[j] & 1 << i)
				connections[j] |= conns;
		}
	}

	// if(connections[__CONN_CHECK_IDX] != __CONN_CHECK)
	// 	return;

	// if(res_idx == 1660128){
	// 	printf("\n\nbx: %d, by: %d, bz: %d, tx: %d, ty: %d, tz: %d\n\n",
	// 			blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);


	// printf("2 - %ld - pts:\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n\n", 
	// res_idx,
	// a[0], a[1], a[2], a[3], 
	// a[4], a[5], a[6], a[7], 
	// a[8], a[9], a[10], a[11],
	// b[0], b[1], b[2], b[3], 
	// b[4], b[5], b[6], b[7], 
	// b[8], b[9], b[10], b[11]);
	// }

	d_res[res_idx] = a[0];
	d_res[res_idx + 1] = a[1];
	d_res[res_idx + 2] = a[2];
	d_res[res_idx + 3] = a[3];
	d_res[res_idx + 4] = a[4];
	d_res[res_idx + 5] = a[5];
	d_res[res_idx + 6] = a[6];
	d_res[res_idx + 7] = a[7];
	d_res[res_idx + 8] = a[8];
	d_res[res_idx + 9] = a[9];
	d_res[res_idx + 10] = a[10];
	d_res[res_idx + 11] = a[11];

	d_res[res_idx + 12] = b[0];
	d_res[res_idx + 13] = b[1];
	d_res[res_idx + 14] = b[2];
	d_res[res_idx + 15] = b[3];
	d_res[res_idx + 16] = b[4];
	d_res[res_idx + 17] = b[5];
	d_res[res_idx + 18] = b[6];
	d_res[res_idx + 19] = b[7];
	d_res[res_idx + 20] = b[8];
	d_res[res_idx + 21] = b[9];
	d_res[res_idx + 22] = b[10];
	d_res[res_idx + 23] = b[11];
	// d_res[res_idx] = maxH - minH;
	// d_res[res_idx + 1] = maxW - minW;
	// d_res[res_idx + 2] = a_idx;
	// d_res[res_idx + 3] = b_idx;
}


int* CudaCombine::createDeviceAdjArray(const std::vector<int>& allReqAdj){
	int* d_adj;
	const size_t mem_size = __SIZE_ADJ * sizeof(int);
	checkCudaErrors(hipMalloc((void **)&d_adj, mem_size));

	int* h_adj = (int*)(allReqAdj.data());
	checkCudaErrors(hipMemcpy(d_adj, h_adj, mem_size, hipMemcpyHostToDevice));

	std::cout << "mem size adj: " << mem_size << ", (MB): " << ((float)mem_size)/1024.0/1024.0 << ", pointer: " << d_adj << std::endl;
	return d_adj;
}


int* CudaCombine::createDeviceConnArray(const std::vector<int>& conns){
	int* d_conn;
	const size_t mem_size = conns.size() * sizeof(int);
	checkCudaErrors(hipMalloc((void **)&d_conn, mem_size));

	int* h_conn = (int*)(conns.data());
	checkCudaErrors(hipMemcpy(d_conn, h_conn, mem_size, hipMemcpyHostToDevice));

	std::cout << "mem size conn: " << mem_size << ", (MB): " << ((float)mem_size)/1024.0/1024.0 << ", pointer: " << d_conn << std::endl;
	return d_conn;
}


int16_t* CudaCombine::createDeviceCoreLayoutsArray(const std::vector<int16_t>& pts){
	int16_t* d_pts;
	const size_t mem_size = pts.size() * sizeof(int16_t);
	checkCudaErrors(hipMalloc((void **)&d_pts, mem_size));

	int16_t* h_pts = (int16_t*)(pts.data());
	checkCudaErrors(hipMemcpy(d_pts, h_pts, mem_size, hipMemcpyHostToDevice));

	std::cout << "mem size core layout: " << mem_size << ", (MB): " << ((float)mem_size)/1024.0/1024.0 << ", pointer: " << d_pts << std::endl;
	return d_pts;
}

int* CudaCombine::createDeviceResArray(const size_t mem_size) {
	int *d_res = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_res, mem_size));
	checkCudaErrors(hipMemset(d_res, -1, mem_size));

	std::cout << "mem size res: " << mem_size << ", (MB): " << ((float)mem_size)/1024.0/1024.0 << ", pointer: " << d_res << std::endl;
	return d_res;
}

void CudaCombine::freeDeviceArrays(int* adj, int* res, int* conn, int16_t* a, int16_t* b) {
	checkCudaErrors(hipFree(a));
	checkCudaErrors(hipFree(b));
	checkCudaErrors(hipFree(adj));
	checkCudaErrors(hipFree(res));
	checkCudaErrors(hipFree(conn));
}

void CudaCombine::createPts(
		const size_t res_mem_size,
		const long NConn,
		const long num_a,
		const long qtd_b,
		const long a_offset,
		const long num_blocks,
		const long num_threads,
		int* h_res,
		int* d_adj,
		int* d_res,
		int* d_conns,
		int16_t* d_a,
		int16_t* d_b) 
	{
	dim3 grid(num_blocks, num_a, NConn);
	dim3 threads(num_threads, 1, 1);

	checkCudaErrors(hipMemset(d_res, -1, res_mem_size));

	k_createPts<<<grid, threads>>>(d_a, d_b, d_res, d_adj, d_conns, num_a, qtd_b, a_offset);
	hipDeviceSynchronize();

	checkCudaErrors(hipMemcpy(h_res, d_res, res_mem_size, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
}