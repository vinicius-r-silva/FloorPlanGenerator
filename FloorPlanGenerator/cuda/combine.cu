#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>
#include <stdint.h>

#include "combine.h"
#include "helper.h"
#include "../lib/cvHelper.h"
#include "../lib/globals.h"

#define __N_A 3
#define __N_B 3
#define __N_PTS 6

#define __SIZE_A 12		// n_a * 4
#define __SIZE_B 12		// n_b * 4
#define __SIZE_PTS 24	// n_pts * 4
#define __SIZE_RES 5	// score, maxH, minH, maxW, minW
#define __SIZE_NBR 9	// __SIZE_A * __SIZE_B conns

#define __LEFT 0
#define __UP 1
#define __RIGHT 2
#define __DOWN 3

// __global__
// void k_checkOverlap(int16_t *d_pts, int16_t *d_res, const uint max_idx){
// 	const int idx = threadIdx.x;
// 	if(idx >= max_idx)
// 		return;

// 	const int pts_idx = idx * __SIZE_PTS;
// 	const int res_idx = idx * __SIZE_RES;

// 	int16_t pts[__SIZE_PTS];
// 	for(int i = 0; i < __SIZE_PTS; i++){
// 		pts[i] = d_pts[pts_idx + i];
// 	}

// 	//left, up, right, down
// 	int16_t notOverlap = 1;
// 	for(int i = 0; i < __SIZE_A && notOverlap; i+=4){
// 		for(int j = __SIZE_A; (j < (__SIZE_A +__SIZE_B)) && notOverlap; j+=4){
// 			// if(idx > 27){
// 			// 	printf("idx: %d, i: %d, j: %d,\ta_down: %d, a_up: %d, a_left: %d, a_right: %d, b_down: %d, b_up: %d, b_left: %d, b_right: %d\n",
// 			// 	idx, i, j,
// 			// 	pts[i + __DOWN], pts[i + __UP], pts[i], pts[i + __RIGHT],
// 			// 	pts[j + __DOWN], pts[j + __UP], pts[j], pts[j + __RIGHT]);
// 			// }

// 			if(((pts[i + __DOWN] > pts[j + __UP] && pts[i + __DOWN] <= pts[j + __DOWN]) ||
// 				(pts[i + __UP]  >= pts[j + __UP] && pts[i + __UP] < pts[j + __DOWN])) &&
// 				((pts[i + __RIGHT] > pts[j] && pts[i + __RIGHT] <= pts[j + __RIGHT]) ||
// 				(pts[i]  >= pts[j] && pts[i]  <  pts[j + __RIGHT]) ||
// 				(pts[i]  <= pts[j] && pts[i + __RIGHT] >= pts[j + __RIGHT]))){
// 					notOverlap = 0;
// 			}

			
// 			else if(((pts[j + __DOWN] > pts[i + __UP] && pts[j + __DOWN] <= pts[i + __DOWN]) ||
// 				(pts[j + __UP] >= pts[i + __UP] && pts[j + __UP] < pts[i + __DOWN])) &&
// 				((pts[j + __RIGHT] > pts[i] && pts[j + __RIGHT] <= pts[i + __RIGHT]) ||
// 				(pts[j]  >= pts[i] && pts[j]  <  pts[i + __RIGHT]) ||
// 				(pts[j]  <= pts[i] && pts[j + __RIGHT] >= pts[i + __RIGHT]))){
// 					notOverlap = 0;
// 			}

			
// 			else if(((pts[i + __RIGHT] > pts[j] && pts[i + __RIGHT] <= pts[j + __RIGHT]) ||
// 				(pts[i] >= pts[j] && pts[i] < pts[j + __RIGHT])) &&
// 				((pts[i + __DOWN] > pts[j + __UP] && pts[i + __DOWN] <= pts[j + __DOWN]) ||
// 				(pts[i + __UP]  >= pts[j + __UP] && pts[i + __UP]   <  pts[j + __DOWN]) ||
// 				(pts[i + __UP]  <= pts[j + __UP] && pts[i + __DOWN] >= pts[j + __DOWN]))){
// 					notOverlap = 0;
// 			}

			
// 			else if(((pts[j + __RIGHT] > pts[i] && pts[j + __RIGHT] <= pts[i + __RIGHT]) ||
// 				(pts[j] >= pts[i] && pts[j] < pts[i + __RIGHT])) &&
// 				((pts[j + __DOWN] > pts[i + __UP] && pts[j + __DOWN] <= pts[i + __DOWN]) ||
// 				(pts[j + __UP]  >= pts[i + __UP] && pts[j + __UP]   <  pts[i + __DOWN]) ||
// 				(pts[j + __UP]  <= pts[i + __UP] && pts[j + __DOWN] >= pts[i + __DOWN]))){
// 					notOverlap = 0;
// 			}
// 		}
// 	}


// 	// if(idx > 27){
// 	// 	printf("idx: %d\t notOverlap: %d\n", idx, notOverlap);
// 	// }
// 	d_res[res_idx] = notOverlap - 1;
// }

// const int num_threads = 768; // 1024
// const int num_blocks = (qtd_b + num_threads -1) / num_threads;
// dim3 grid(num_blocks, num_a, NConn);
// dim3 threads(num_threads, 1, 1);

__global__ 
void k_createPts(int16_t *d_a, int16_t *d_b, int16_t *d_res, int8_t *d_nbr, const int qtd_a, const int qtd_b, const int a_offset) {
	const int k = blockIdx.z + 1 + blockIdx.z/4;
	const int a_idx = (blockIdx.y + a_offset) * __SIZE_A;
	const int b_idx = blockIdx.x * blockDim.x + threadIdx.x;
	// int res_idx = ((blockIdx.z * qtd_a + blockIdx.y) * qtd_b +  b_idx) * __SIZE_PTS;
	int res_idx = ((blockIdx.z * qtd_a + blockIdx.y) * qtd_b +  b_idx) * __SIZE_RES;
	const int nbr_idx = ((blockIdx.z * qtd_a + blockIdx.y) * qtd_b +  b_idx) * __SIZE_NBR;

	if(b_idx >= qtd_b || blockIdx.y >= qtd_a)
		return;


	__shared__ int16_t a[__SIZE_A];
	if(threadIdx.x < __SIZE_A){
		a[threadIdx.x] = d_a[a_idx + threadIdx.x];
	}
  	__syncthreads();


	int16_t b[__SIZE_B];
	for(int i = 0; i < __SIZE_B; i++){
		b[i] = d_b[b_idx*__SIZE_B + i];
	}

	int8_t Neighbors[__SIZE_NBR];
	
	const int srcConn = k & 0b11;
	const int dstConn = (k >> 2) & 0b11;
	
	int dst = 0;
	int src = 0;
	if(dstConn == 0 || dstConn == 2)
		dst = b[0];
	else 
		dst = b[2];

	if(srcConn == 0 || srcConn == 2)
		src = a[__SIZE_A - 4];
	else 
		src = a[__SIZE_A - 2];

	const int diffX = src - dst;
	for(int i = 0; i < __SIZE_B; i+=2){
		b[i] += diffX;
	}

	if(dstConn == 0 || dstConn == 1)
		dst = b[1];
	else 
		dst = b[3];
		
	if(srcConn == 0 || srcConn == 1)
		src = a[__SIZE_A - 3];
	else 
		src = a[__SIZE_A - 1];

	const int diffY = src - dst;
	for(int i = 1; i < __SIZE_B; i+=2){
		b[i] += diffY;
	}

	// for(int i = 0; i < __SIZE_A; i++){
	// 	d_pts[res_idx + i] = a[i];
	// }

	int16_t minH = 5000, maxH = -5000;
	int16_t minW = 5000, maxW = -5000;
	for(int i = 0; i < __SIZE_B; i+=4){
		if(b[i + __UP] < minH)
			maxH = b[i + __UP];
		if(b[i + __DOWN] > maxH)
			maxH = b[i + __DOWN];
		if(b[i] < minW)
			minW = b[i];
		if(b[i + __RIGHT] > maxW)
			maxW = b[i + __RIGHT];
	}

	// res_idx += __SIZE_A;
	// for(int i = 0; i < __SIZE_B; i++){
	// 	d_pts[res_idx + i] = b[i];
	// }

	//left, up, right, down
	int8_t notOverlap = 1;
	for(int i = 0; i < __SIZE_A && notOverlap; i+=4){
		const int a_left = a[i];
		const int a_up = a[i + __UP];
		const int a_down = a[i + __DOWN];
		const int a_right = a[i + __RIGHT];

		if(a_up < minH)
			maxH = a_up;
		if(a_down > maxH)
			maxH = a_down;
		if(a_left < minW)
			minW = a_left;
		if(a_right > maxW)
			maxW = a_right;

		for(int j = 0; j < __SIZE_B && notOverlap; j+=4){
			const int b_left = b[j];
			const int b_up = b[j + __UP];
			const int b_down = b[j + __DOWN];
			const int b_right = b[j + __RIGHT];
			// if(idx > 27){
			// 	printf("idx: %d, i: %d, j: %d,\ta_down: %d, a_up: %d, a_left: %d, a_right: %d, b_down: %d, b_up: %d, b_left: %d, b_right: %d\n",
			// 	idx, i, j,
			// 	pts[i + __DOWN], pts[i + __UP], pts[i], pts[i + __RIGHT],
			// 	pts[j + __DOWN], pts[j + __UP], pts[j], pts[j + __RIGHT]);
			// }

			if(((a_down > b_up && a_down <= b_down) ||
				(a_up  >= b_up && a_up < b_down)) &&
				((a_right > b_left && a_right <= b_right) ||
				(a_left  >= b_left && a_left  <  b_right) ||
				(a_left  <= b_left && a_right >= b_right))){
					notOverlap = 0;
			}

			
			else if(((b_down > a_up && b_down <= a_down) ||
				(b_up >= a_up && b_up < a_down)) &&
				((b_right > a_left && b_right <= a_right) ||
				(b_left  >= a_left && b_left  <  a_right) ||
				(b_left  <= a_left && b_right >= a_right))){
					notOverlap = 0;
			}

			
			else if(((a_right > b_left && a_right <= b_right) ||
				(a_left >= b_left && a_left < b_right)) &&
				((a_down > b_up && a_down <= b_down) ||
				(a_up  >= b_up && a_up   <  b_down) ||
				(a_up  <= b_up && a_down >= b_down))){
					notOverlap = 0;
			}

			
			else if(((b_right > a_left && b_right <= a_right) ||
				(b_left >= a_left && b_left < a_right)) &&
				((b_down > a_up && b_down <= a_down) ||
				(b_up  >= a_up && b_up   <  a_down) ||
				(b_up  <= a_up && b_down >= a_down))){
					notOverlap = 0;
			}

			// if(!notOverlap)
			// 	break;

			const int i_id = i / 4;
			const int j_id = j / 4;
			Neighbors[i_id * __N_B + j_id] = 0;
			if((a_down == b_up || a_up == b_down) && 
			  ((a_left > b_left && a_left < b_right) || 
			   (b_left > a_left && b_left < a_right) ||
			   (a_right < b_right && a_right > b_left) || 
			   (b_right < a_right && b_right > a_left)))
					Neighbors[i_id * __N_B + j_id] = 1;

			if((a_left == b_right || a_right == b_left) && 
			  ((a_up > b_up && a_up < b_down) || 
			   (b_up > a_up && b_up < a_down) ||
			   (a_down < b_down && a_down > b_up) || 
			   (b_down < a_down && b_down > a_up)))
					Neighbors[i_id * __N_B + j_id] = 1;

		}
	}

	d_res[res_idx] = notOverlap - 1;
	if(!notOverlap){
		return;
	}

	d_res[res_idx + 1] = maxH;
	d_res[res_idx + 2] = maxW;
	d_res[res_idx + 3] = minH;
	d_res[res_idx + 4] = minW;

	// for(int i = 0; i < __SIZE_NBR; i++){
	// 	d_nbr[i + nbr_idx] = Neighbors[i];
	// }
	// printf("a_idx: %d,\tb_idx: %d,\tres_idx: %d,\tblockIdx.x: %d,\tblockIdx.y: %d,\tblockIdx.z: %d,\tdblockDim.x: %d,\tthreadIdx.x: %d\n", a_idx, b_idx, res_idx, blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x);
	// printf("a_idx: %d,\tb_idx: %d,\tres_idx: %d,\tblockIdx.x: %d,\tblockIdx.y: %d,\tblockIdx.z: %d,\tdblockDim.x: %d,\tthreadIdx.x: %d,\tdiffX: %d,\tdiffY: %d,\ta[0]: %d,\ta[1]: %d\n", a_idx, b_idx, res_idx - __SIZE_A, blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, diffX, diffY, a[0], a[1]);
}

void gpuHandler::createPts(const std::vector<int16_t>& a, const std::vector<int16_t>& b) {
	const int NConn = 12;
	const long num_a = 200;
	const int qtd_a = a.size() / __SIZE_A;
	const int qtd_b = b.size() / __SIZE_B;

	// const long NConn = 2;
	// const long qtd_a = 2; 
	// const long qtd_b = 12; //minimum 12
	// const long num_a = qtd_a;

	findCudaDevice();	

	const long aLayoutSize = sizeof(int16_t) * __SIZE_A;
	const long bLayoutSize = sizeof(int16_t) * __SIZE_B;
	// const long ptsLayoutSize = sizeof(int16_t) * __SIZE_PTS;
	const long resLayoutSize = sizeof(int16_t) * __SIZE_RES;
	const long nbrLayoutSize = sizeof(int8_t) * __SIZE_NBR;
	const unsigned long mem_size_a = aLayoutSize * qtd_a;
	const unsigned long mem_size_b = bLayoutSize * qtd_b;
	// const unsigned long mem_size_pts = num_a * NConn * qtd_b * ptsLayoutSize;
	const unsigned long mem_size_res = num_a * NConn * qtd_b * resLayoutSize;
	const unsigned long mem_size_nbr = num_a * NConn * qtd_b * nbrLayoutSize;
	
	// allocate host memory
	int16_t *h_a = (int16_t *)(&a[0]);
	int16_t *h_b = (int16_t *)(&b[0]);
	// int16_t *h_pts = (int16_t *)malloc(mem_size_pts);
	// int16_t *h_res = (int16_t *)malloc(mem_size_res);
	// int8_t *h_nbr = (int8_t *)malloc(mem_size_nbr);
 	// int8_t *h_nbr = nullptr;
	int16_t *h_res = nullptr;

	// hipHostMalloc((void**)&h_nbr, mem_size_nbr);
	hipHostMalloc((void**)&h_res, mem_size_res);


	// Allocate CUDA events that we'll use for timing
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	// setup execution parameters
	const int num_threads = 768; // 1024
	const int num_blocks = (qtd_b + num_threads -1) / num_threads;

	dim3 grid(num_blocks, num_a, NConn);
	dim3 threads(num_threads, 1, 1);

	// allocate device memory
	// int16_t *d_a, *d_b, *d_pts, *d_res;
	int16_t *d_a, *d_b, *d_res;
	int8_t *d_nbr;
	checkCudaErrors(hipMalloc((void **)&d_a, mem_size_a));
	checkCudaErrors(hipMalloc((void **)&d_b, mem_size_b));
	// checkCudaErrors(hipMalloc((void **)&d_pts, mem_size_pts));
	checkCudaErrors(hipMalloc((void **)&d_res, mem_size_res));
	checkCudaErrors(hipMalloc((void **)&d_nbr, mem_size_nbr));

	// copy host data to device
  	checkCudaErrors(hipEventRecord(start));
	checkCudaErrors(hipMemcpy(d_a, h_a, mem_size_a, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_b, h_b, mem_size_b, hipMemcpyHostToDevice));

	for(int i = 0; i < qtd_a; i += num_a){
		int diff = qtd_a - i;
		if(diff < num_a){
			k_createPts<<<grid, threads>>>(d_a, d_b, d_res, d_nbr, diff, qtd_b, i);
			hipMemcpy(h_res, d_res, mem_size_res, hipMemcpyDeviceToHost);
		} else {
			k_createPts<<<grid, threads>>>(d_a, d_b, d_res, d_nbr, num_a, qtd_b, i);
			hipMemcpy(h_res, d_res, mem_size_res, hipMemcpyDeviceToHost);
		}
	}

	// k_createPts<<<grid, threads>>>(d_a, d_b, d_pts, num_a, qtd_b, 0);
	// k_createPts<<<grid, threads>>>(d_a, d_b, d_res, num_a, qtd_b, 0);
	// k_checkOverlap<<<1, num_threads>>>(d_pts, d_res, qtd_b);

	// hipDeviceSynchronize();	
  	checkCudaErrors(hipEventRecord(stop));
  	checkCudaErrors(hipEventSynchronize(stop));

  	float msecTotal = 0.0f;
  	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

	// std::cout << "mem_size_a: " << mem_size_a << ", mem_size_b: " << mem_size_b << ", mem_size_pts: " << mem_size_pts << ", mem_size_res: " << mem_size_res << std::endl;
	// std::cout << "mem_size_a (MB): " << ((float)mem_size_a)/1024.0/1024.0 << ", mem_size_b (MB): " << ((float)mem_size_b)/1024.0/1024.0 << ", mem_size_pts (MB): " << ((float)mem_size_pts)/1024.0/1024.0 << ", mem_size_res (MB): " << ((float)mem_size_res)/1024.0/1024.0 << std::endl;
	std::cout << "mem_size_a: " << mem_size_a << ", mem_size_b: " << mem_size_b << ", mem_size_res: " << mem_size_res << ", mem_size_nbr: " << mem_size_nbr << std::endl;
	std::cout << "mem_size_a (MB): " << ((float)mem_size_a)/1024.0/1024.0 << ", mem_size_b (MB): " << ((float)mem_size_b)/1024.0/1024.0 << ", mem_size_res (MB): " << ((float)mem_size_res)/1024.0/1024.0 << ", mem_size_nbr (MB): " << ((float)mem_size_nbr)/1024.0/1024.0 << std::endl;

	std::cout << "a.size(): " << a.size() << ", b.size(): " << b.size() << std::endl;
	std::cout << "qtd_a: " << qtd_a << ", qtd_b: " << qtd_b << std::endl;

	std::cout << "num_threads: " << num_threads << ", num_blocks: " << num_blocks << std::endl;
	std::cout << "grid: " << grid.x << ", " << grid.y << ", " << grid.z << std::endl;
	std::cout << "threads: " << threads.x << ", " << threads.y << ", " << threads.z << std::endl;
	std::cout << "Time: " << msecTotal << std::endl;

	// check if kernel execution generated and error
	getLastCudaError("Kernel execution failed");

	// copy results from device to host
	// checkCudaErrors(hipMemcpy(h_pts, d_pts, mem_size_pts, hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(h_res, d_res, mem_size_res, hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(h_nbr, d_nbr, mem_size_nbr, hipMemcpyDeviceToHost));

	// cleanup device memory
	checkCudaErrors(hipFree(d_a));
	checkCudaErrors(hipFree(d_b));
	// checkCudaErrors(hipFree(d_pts));
	checkCudaErrors(hipFree(d_res));


	// std::cout << "A: " << std::endl;
	// for(int i = 0; i < qtd_a * __SIZE_A; i+=__SIZE_A){
	// 	for(int j = 0; j < __SIZE_A; j++){
	// 		std::cout << h_a[i + j] << ", ";
	// 	}
	// 	std::cout << std::endl;
	// }

	// std::cout << std::endl << "B: " << std::endl;
	// for(int i = 0; i < qtd_b * __SIZE_B; i+=__SIZE_B){
	// 	for(int j = 0; j < __SIZE_B; j++){
	// 		std::cout << h_b[i + j] << ", ";
	// 	}
	// 	std::cout << std::endl;
	// }

	// std::cout << std::endl << "Res: " << std::endl;
	// for(int i = 0; i < NConn; i++){
	// 	for(int j = 0; j < qtd_a; j++){
	// 		for(int k = 0; k < qtd_b; k++){
	// 			int baseIdx = ((i * qtd_a + j) * qtd_b + k) * __SIZE_PTS;
	// 			std::cout << "NConn: " << i << ", a: " << j << ", b: " << k << ", idx: " << baseIdx << " :     ";
	// 			for(int l = 0; l < __SIZE_PTS; l++, baseIdx++){
	// 				std::cout << h_pts[baseIdx] << ", ";
	// 			}
	// 			std::cout << std::endl;
	// 		}
	// 	}
	// }

// #ifdef OPENCV_ENABLED 
// 	int i = 0;
// 	const int max_i = (mem_size_pts / sizeof(int16_t))  - __SIZE_PTS;
// 	std::vector<int16_t> PtsX(__SIZE_PTS/2, 0);
// 	std::vector<int16_t> PtsY(__SIZE_PTS/2, 0);
// 	while(i <= max_i){
// 		for(int j = 0; j < __SIZE_PTS; j+=2){
// 			PtsX[j/2] = h_pts[i + j];
// 			PtsY[j/2] = h_pts[i + j + 1];
// 		}
		
// 		std::cout << "i: " << i << ", i_idx: " << i / __SIZE_PTS << ", res_idx: " << (i / __SIZE_PTS) * __SIZE_RES << ", res: " << h_res[(i / __SIZE_PTS) * __SIZE_RES] << std::endl;
// 		int c = CVHelper::showLayoutMove(PtsX, PtsY);
// 		// i +=__SIZE_PTS * qtd_b * c;
// 		i += __SIZE_PTS*c;
// 		if(i < 0)
// 			i = 0;
// 	}
// #endif


	// cleanup host memory
	// free(h_pts);
	// free(h_res);
	checkCudaErrors(hipFree(h_res));
}