#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include "helloWorld.h"

__global__ void printHelloGPU()
{
	printf("Hello World from the GPUn");
}

int Cuda_test::launchGPU()
{
	printHelloGPU<<<1, 1>>>();
	hipDeviceSynchronize();
	
	// getchar();
	return 0;
}