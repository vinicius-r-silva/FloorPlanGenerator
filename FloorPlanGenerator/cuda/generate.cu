#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>
#include <stdint.h>
#include <algorithm>
#include <filesystem>
#include <fstream>

#include "helper.h"
#include "generate.h"
#include "process.h"
#include "../lib/log.h"
#include "../lib/cvHelper.h"
#include "../lib/globals.h"
#include "../lib/calculator.h"


// __global__
// void createSizes(int *d_rooms_config, int8_t *d_room_sizes, const long offset, const long maxIdx){
// 	long idx = blockIdx.x * blockDim.x + threadIdx.x;
// 	if(idx > maxIdx)
// 		return;
	
// 	const long res_idx = idx * __GENERATE_SIZE_LENGHT * __GENERATE_ROTATIONS;
// 	idx += offset;

// 	if(res_idx < 0){
// 		printf("aa");
// 		return;
// 	}
// 	if(idx < 0){
// 		printf("bb");
// 		return;
// 	}

// 	__shared__ int rooms_config[__GENERATE_N * __ROOM_CONFIG_LENGHT];
// 	if(threadIdx.x < __GENERATE_N * __ROOM_CONFIG_LENGHT){
// 		rooms_config[threadIdx.x] = d_rooms_config[threadIdx.x];
// 	}

// 	// //TODO remove this for sizes < 30
// 	// if(maxIdx < __GENERATE_N * __ROOM_CONFIG_LENGHT && threadIdx.x == 0){
// 	// 	for(int i = 0; i < __GENERATE_N * __ROOM_CONFIG_LENGHT; i++){
// 	// 		rooms_config[i] = d_rooms_config[i];
// 	// 		// printf("%d, ", rooms_config[i]);
// 	// 	}
// 	// 	// printf("\n");
// 	// }

// 	__syncthreads();

// 	int8_t room_sizes[__GENERATE_SIZE_LENGHT];
// 	int validRotations[__GENERATE_N];
// 	for(int i = 0; i < __GENERATE_N; i++){
// 		validRotations[i] = 0;
// 	}

// 	for(int i = 0; i < __GENERATE_N; i++){
// 		const int sizes_idx = i * 2;
// 		const int room_idx = i * __ROOM_CONFIG_LENGHT;
// 		const int step = rooms_config[room_idx + __ROOM_CONFIG_STEP];
// 		const int minH = rooms_config[room_idx + __ROOM_CONFIG_MINH];
// 		const int maxH = rooms_config[room_idx + __ROOM_CONFIG_MAXH];
// 		const int minW = rooms_config[room_idx + __ROOM_CONFIG_MINW];
// 		const int maxW = rooms_config[room_idx + __ROOM_CONFIG_MAXW];
// 		const int countH = rooms_config[room_idx + __ROOM_CONFIG_COUNTH];
// 		const int countW = rooms_config[room_idx + __ROOM_CONFIG_COUNTW];

// 		room_sizes[sizes_idx] = ((idx % countH) * step) + minH;
// 		idx /= countH;

// 		room_sizes[sizes_idx + 1] = ((idx % countW) * step) + minW;
// 		idx /= countW;

// 		validRotations[i] = room_sizes[sizes_idx] < minW || room_sizes[sizes_idx] > maxW;

// 		if(room_sizes[sizes_idx] > maxH){
// 			room_sizes[sizes_idx] = maxH;
// 		};
		
// 		if(room_sizes[sizes_idx + 1] > minW){
// 			room_sizes[sizes_idx + 1] = minW;
// 		};
// 	}

// 	for(int i = 0; i < __GENERATE_ROTATIONS; i++){
// 		int validRotation = 1;
// 		for(int j = 0; j < __GENERATE_N && validRotation; j++){
// 			if(i & (1 << j)){
// 				validRotation = validRotations[j];
// 			}
// 		}

// 		if(!validRotation)
// 			continue;


// 		const long offset = res_idx + (i * __GENERATE_SIZE_LENGHT);
// 		for(int j = 0; j < __GENERATE_N; j++){
// 			if(i & (1 << j)){
// 				d_room_sizes[offset + (j * 2)] = room_sizes[(j * 2) + 1];
// 				d_room_sizes[offset + (j * 2) + 1] = room_sizes[(j * 2)];
// 			} else {
// 				d_room_sizes[offset + (j * 2)] = room_sizes[(j * 2)];
// 				d_room_sizes[offset + (j * 2) + 1] = room_sizes[(j * 2) + 1];
// 			}
// 		}
// 	}
// }


// int8_t* CudaGenerate::generateSizes(int* d_configs, const long qtdSizes){
// 	int8_t *d_sizes = nullptr;
// 	const long sizes_mem_size = qtdSizes * __GENERATE_SIZE_LENGHT * __GENERATE_ROTATIONS * sizeof(int8_t);
// 	hipMalloc((void**)&d_sizes, sizes_mem_size);	
// 	checkCudaErrors(hipMemset(d_sizes, 0, sizes_mem_size));

// 	std::cout << "sizes_mem_size: " << sizes_mem_size << ", Mb: " << ((double)sizes_mem_size) / 1024.0 / 1024.0 << std::endl;

// 	const int threadsPerBlock = qtdSizes > 768 ? 768 : qtdSizes;
// 	const int qtdBlocks = (qtdSizes + threadsPerBlock - 1) / threadsPerBlock;

// 	dim3 grid(qtdBlocks, 1, 1);
// 	dim3 threads(threadsPerBlock, 1, 1);

// 	createSizes<<<grid, threads>>>(d_configs, d_sizes, 0, qtdSizes);
// 	hipDeviceSynchronize();	

// 	// int8_t *h_sizes = nullptr;
// 	// hipHostMalloc((void**)&h_sizes, sizes_mem_size);	
// 	// checkCudaErrors(hipMemcpy(h_sizes, d_sizes, sizes_mem_size, hipMemcpyDeviceToHost));
// 	// hipDeviceSynchronize();	

// 	// for(int i = 0; i < qtdSizes * __GENERATE_ROTATIONS; i++){
// 	// 	for(int j = 0; j < __GENERATE_SIZE_LENGHT; j++){
// 	// 		std::cout << (int)h_sizes[(i * __GENERATE_SIZE_LENGHT) + j] << ", ";
// 	// 	}
// 	// 	std::cout << std::endl;
// 	// }
// 	// std::cout << std::endl;

// 	return d_sizes;
// }


__global__
void generate(int *d_rooms_config, int *d_perm, int16_t *d_res, const long size_idx_offset, const long max_size_idx){
	// const int conn_idx = blockIdx.y;
	const int perm_idx = blockIdx.z;
	const int rotation_idx = threadIdx.y;
	long size_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const long res_idx = ((blockIdx.z * gridDim.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y)  + (threadIdx.y * blockDim.x) + threadIdx.x) * (long)__GENERATE_RES_LENGHT;

	if(size_idx > max_size_idx)
		return;

	size_idx += size_idx_offset;

	__shared__ int rooms_config[__GENERATE_N * __ROOM_CONFIG_LENGHT];
	if(threadIdx.x < (__GENERATE_N * __ROOM_CONFIG_LENGHT) && threadIdx.y == 0){
		rooms_config[threadIdx.x] = d_rooms_config[threadIdx.x];
	}

	__shared__ int perm[__GENERATE_N * __GENERATE_PERM];
	if(threadIdx.x < (__GENERATE_N * __GENERATE_PERM) && threadIdx.y == 0){
		perm[threadIdx.x] = d_perm[threadIdx.x];
	}

	__syncthreads();


	int result[__GENERATE_RES_LAYOUT_LENGHT];
	for(int i = 0; i < __GENERATE_RES_LAYOUT_LENGHT; i++){
		result[i] = 0;
	}

	for(int i = 0; i < __GENERATE_N; i++){
		const int id = perm[(perm_idx * __GENERATE_N) + i];
		const int offset_idx = (i * 4) + 2;
		const int room_idx = id * __ROOM_CONFIG_LENGHT;
		const int step = rooms_config[room_idx + __ROOM_CONFIG_STEP];
		const int minH = rooms_config[room_idx + __ROOM_CONFIG_MINH];
		const int maxH = rooms_config[room_idx + __ROOM_CONFIG_MAXH];
		const int minW = rooms_config[room_idx + __ROOM_CONFIG_MINW];
		const int maxW = rooms_config[room_idx + __ROOM_CONFIG_MAXW];
		const int countH = rooms_config[room_idx + __ROOM_CONFIG_COUNTH];
		const int countW = rooms_config[room_idx + __ROOM_CONFIG_COUNTW];

		int h = ((size_idx % countH) * step) + minH;
		if(h > maxH){
			h = maxH;
		}
		size_idx /= countH;

		int w = ((size_idx % countW) * step) + minW;
		if(w > maxW){
			w = maxW;
		}
		size_idx /= countW;

		if(rotation_idx & (1 << id)){
			if(w == h)
				return;

			result[offset_idx] = w;
			result[offset_idx + 1] = h;
		} else {
			result[offset_idx] = h;
			result[offset_idx + 1] = w;
		}
	}

	for(int i = 0; i < __GENERATE_RES_LAYOUT_LENGHT; i++){
		d_res[res_idx + i] = result[i];
	}
	d_res[res_idx + __GENERATE_RES_LAYOUT_LENGHT] = perm_idx;


}

int* CudaGenerate::createDeviceRoomConfigsArray(const std::vector<RoomConfig>& rooms){
	const long configs_mem_size = __GENERATE_N * __ROOM_CONFIG_LENGHT * sizeof(int);
	
	int *h_configs = nullptr;
	hipHostMalloc((void**)&h_configs, configs_mem_size);	
	
	for(int i = 0; i < __GENERATE_N; i++){
		const int offset = i * __ROOM_CONFIG_LENGHT;
		h_configs[offset + __ROOM_CONFIG_STEP] = rooms[i].step;
		h_configs[offset + __ROOM_CONFIG_MINH] = rooms[i].minH;
		h_configs[offset + __ROOM_CONFIG_MAXH] = rooms[i].maxH;
		h_configs[offset + __ROOM_CONFIG_MINW] = rooms[i].minW;
		h_configs[offset + __ROOM_CONFIG_MAXW] = rooms[i].maxW;

		const int countH = (((rooms[i].maxH - rooms[i].minH) + rooms[i].step - 1) / rooms[i].step) + 1;
		const int countW = (((rooms[i].maxW - rooms[i].minW) + rooms[i].step - 1) / rooms[i].step) + 1;
		h_configs[offset + __ROOM_CONFIG_COUNTH] = countH;
		h_configs[offset + __ROOM_CONFIG_COUNTW] = countW;
	}

	int *d_configs = nullptr;
	checkCudaErrors(hipMalloc((void **)&d_configs, configs_mem_size));
	checkCudaErrors(hipMemcpy(d_configs, h_configs, configs_mem_size, hipMemcpyHostToDevice));
	hipDeviceSynchronize();	

	checkCudaErrors(hipHostFree(h_configs));
	return d_configs;
}

int* CudaGenerate::createDevicePermArray(){
	const long perm_mem_size = __GENERATE_N * __GENERATE_PERM * sizeof(int);

	int *h_perm = nullptr;
	hipHostMalloc((void**)&h_perm, perm_mem_size);	
	
	std::vector<int> perm;
	for(int i = 0; i < __GENERATE_N; i++){
		perm.push_back(i);
	}

	int idx = 0;
	do {
		for(int i = 0; i < __GENERATE_N; i++){
			h_perm[(idx * __GENERATE_N) + i] = perm[i];
		}
		idx++;
	} while (std::next_permutation(perm.begin(), perm.end()));
	std::cout << std::endl;


	for(int i = 0; i < __GENERATE_PERM; i++){
		std::cout << "perm " << i << ": ";
		for(int j = 0; j < __GENERATE_N; j++){
			std::cout << h_perm[(i * __GENERATE_N) + j] << ", ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;

	int *d_perm = nullptr;
	checkCudaErrors(hipMalloc((void **)&d_perm, perm_mem_size));
	checkCudaErrors(hipMemcpy(d_perm, h_perm, perm_mem_size, hipMemcpyHostToDevice));
	hipDeviceSynchronize();	

	checkCudaErrors(hipHostFree(h_perm));
	return d_perm;
}
 
void CudaGenerate::generateCuda(const std::vector<RoomConfig>& rooms) {
	if(rooms.size() != __GENERATE_N)
		return;

	std::cout << std::endl << std::endl << std::endl;
	for(int i = 0; i < __GENERATE_N; i++){
		Log::print(rooms[i]);
	}

	// const long targetMemSize = (45l * 1024l * 1024l * 1024l) / 10l;
	const long targetMemSize = 8l * 1024l * 1024l * 1024l;

	long NSizes = 1;
    for(const RoomConfig room : rooms){
		NSizes *= (((room.maxH - room.minH + room.step - 1) / room.step) + 1) * (((room.maxW - room.minW + room.step - 1) / room.step) + 1);
    }

    const long NConn = Calculator::NConnections(__GENERATE_N);
    const long NPerm = Calculator::Factorial(__GENERATE_N);
    const long NSizesRotation = NSizes * __GENERATE_ROTATIONS;

	std::cout << "NConn: " << NConn << ", NPerm: " << NPerm << std::endl;
	std::cout << "NSizes: " << NSizes << ", NSizesRotation: " << NSizesRotation << std::endl;

	const int targetThreadsPerBlock = 768;
	const int targetQtdThreadsX = targetThreadsPerBlock / __GENERATE_ROTATIONS;
	if(targetThreadsPerBlock % __GENERATE_ROTATIONS != 0){
		std::cout << "!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!" << std::endl;
		std::cout << "!!!!!!!!!!!!!!!! make the targetThreadsPerBlock divisible by " << __GENERATE_ROTATIONS << "!!!!!!!!!!!!!!!!" << std::endl;
		std::cout << "!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!" << std::endl;
		return; 
	}

	if(targetThreadsPerBlock < __GENERATE_N * __ROOM_CONFIG_LENGHT){
		std::cout << "!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!" << std::endl;
		std::cout << "!!!!!!!!!!!!!!!! not enought threads to fill config array !!!!!!!!!!!!!!!!" << std::endl;
		std::cout << "!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!" << std::endl;
		return; 
	}

	

	const long maxLayoutsPerKernel = targetMemSize / (__GENERATE_RES_LENGHT * sizeof(int16_t));
	const long maxQtdSizes = (maxLayoutsPerKernel / (NConn * NPerm * targetQtdThreadsX * __GENERATE_ROTATIONS)) * targetQtdThreadsX;
	const long qtdSizes = maxQtdSizes < NSizes ? maxQtdSizes : NSizes;
	const long layoutsPerKernel = qtdSizes * NConn * NPerm * __GENERATE_ROTATIONS;

	std::cout << "maxLayoutsPerKernel: " << maxLayoutsPerKernel << std::endl;
	std::cout << "qtdSingleSize: " << NConn * NPerm << std::endl;
	std::cout << "layoutsPerKernel: " << layoutsPerKernel << std::endl;
	std::cout << "maxQtdSizes: " << maxQtdSizes << ", qtdSizes: " << qtdSizes << std::endl;
	std::cout << "kernel launchs: " << NConn * NPerm * (qtdSizes / targetQtdThreadsX)  << std::endl;

	int* d_configs = CudaGenerate::createDeviceRoomConfigsArray(rooms);
	int* d_perm = CudaGenerate::createDevicePermArray();

	int16_t *d_res = nullptr;
	const long result_mem_size = qtdSizes * NConn * NPerm * __GENERATE_ROTATIONS * __GENERATE_RES_LENGHT * sizeof(int16_t);

	hipMalloc((void**)&d_res, result_mem_size);	
	checkCudaErrors(hipMemset(d_res, 0, result_mem_size));

	const int qtdThreadX = qtdSizes > targetQtdThreadsX ? targetQtdThreadsX : qtdSizes;
	const int qtdBlocksX = (qtdSizes + qtdThreadX - 1) / qtdThreadX;

	dim3 grid(qtdBlocksX, NConn, NPerm);
	dim3 threads(qtdThreadX, __GENERATE_ROTATIONS, 1);

	std::cout << "result_mem_size: " << result_mem_size << std::endl;
	std::cout << "targetThreadsPerBlock: " << targetThreadsPerBlock << ", targetQtdThreadsX: " << targetQtdThreadsX << std::endl;
	std::cout << "qtdThreadX: " << qtdThreadX << ", qtdBlocksX: " << qtdBlocksX << std::endl;
	std::cout << "grid: " << grid.x << ", " << grid.y << ", " << grid.z << std::endl;
	std::cout << "threads: " << threads.x << ", " << threads.y << ", " << threads.z << std::endl;

	generate<<<grid, threads>>>(d_configs, d_perm, d_res, 0, qtdSizes);
	hipDeviceSynchronize();	
	// for(int i = 0; i < NSizes; i+= qtdSizes){
	// 	int diff = NSizes - i;

	// 	if(diff < qtdSizes){
	// 		generate<<<grid, threads>>>(d_configs, d_res, i, diff);
	// 		hipDeviceSynchronize();	
	// 	} else {
	// 		generate<<<grid, threads>>>(d_configs, d_res, i, qtdSizes);
	// 		hipDeviceSynchronize();	
	// 	}
	// }

	int16_t *h_res = nullptr;
	hipHostMalloc((void**)&h_res, result_mem_size);	
	checkCudaErrors(hipMemcpy(h_res, d_res, result_mem_size, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();	

	for(int i = 0; i < layoutsPerKernel; i++){
	// for(int i = 0; i < layoutsPerKernel; i+= qtdBlocksX * qtdThreadX * NConn * __GENERATE_ROTATIONS){
		// if(h_res[(i * __GENERATE_RES_LENGHT) + 2] == 0)
		// 	continue;

		std::cout << i * __GENERATE_RES_LENGHT << ":  ";
		for(int j = 0; j < __GENERATE_RES_LENGHT; j++){
			std::cout << (int)h_res[(i * __GENERATE_RES_LENGHT) + j] << ", ";
		}
		std::cout << std::endl;
		getchar();
	}
	std::cout << std::endl;
	checkCudaErrors(hipHostFree(h_res));

	checkCudaErrors(hipFree(d_configs));
	checkCudaErrors(hipFree(d_res));
}