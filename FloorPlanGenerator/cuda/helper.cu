#include <cstdio>
#include <vector>
#include <stdint.h>
#include "helper.h"

CudaHelper::CudaHelper(){
}
 
template <typename T>
void CudaHelper::check(T result, char const *const func, const char *const file, int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}

// Beginning of GPU Architecture definitions
int CudaHelper::_ConvertSMVer2Cores(int major, int minor) {
  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {0x80,  64},
      {0x86, 128},
      {0x87, 128},
      {0x90, 128},
      {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  // If we don't find the values, we default use the previous one
  // to run properly
  printf(
      "MapSMtoCores for SM %d.%d is undefined."
      "  Default to use %d Cores/SM\n",
      major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

// This function returns the best GPU (with maximum GFLOPS)
int CudaHelper::gpuGetMaxGflopsDeviceId() {
  int current_device = 0, sm_per_multiproc = 0;
  int max_perf_device = 0;
  int device_count = 0;
  int devices_prohibited = 0;

  uint64_t max_compute_perf = 0;
  checkCudaErrors(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }

  // Find the best CUDA capable GPU device
  current_device = 0;

  while (current_device < device_count) {
    int computeMode = -1, major = 0, minor = 0;
    checkCudaErrors(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, current_device));

    // If this GPU is not running on Compute Mode prohibited,
    // then we can add it to the list
    if (computeMode != hipComputeModeProhibited) {
      if (major == 9999 && minor == 9999) {
        sm_per_multiproc = 1;
      } else {
        sm_per_multiproc =
            _ConvertSMVer2Cores(major,  minor);
      }
      int multiProcessorCount = 0, clockRate = 0;
      checkCudaErrors(hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, current_device));
      hipError_t result = hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, current_device);
      if (result != hipSuccess) {
        // If hipDeviceAttributeClockRate attribute is not supported we
        // set clockRate as 1, to consider GPU with most SMs and CUDA Cores.
        if(result == hipErrorInvalidValue) {
          clockRate = 1;
        }
        else {
          fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \n", __FILE__, __LINE__,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result));
          exit(EXIT_FAILURE);
        }
      }
      uint64_t compute_perf = (uint64_t)multiProcessorCount * sm_per_multiproc * clockRate;

      if (compute_perf > max_compute_perf) {
        max_compute_perf = compute_perf;
        max_perf_device = current_device;
      }
    } else {
      devices_prohibited++;
    }

    ++current_device;
  }

  if (devices_prohibited == device_count) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " all devices have compute mode prohibited.\n");
    exit(EXIT_FAILURE);
  }

  return max_perf_device;
}



const char* CudaHelper::_ConvertSMVer2ArchName(int major, int minor) {
  // Defines for GPU Architecture types (using the SM version to determine
  // the GPU Arch name)
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    const char* name;
  } sSMtoArchName;

  sSMtoArchName nGpuArchNameSM[] = {
      {0x30, "Kepler"},
      {0x32, "Kepler"},
      {0x35, "Kepler"},
      {0x37, "Kepler"},
      {0x50, "Maxwell"},
      {0x52, "Maxwell"},
      {0x53, "Maxwell"},
      {0x60, "Pascal"},
      {0x61, "Pascal"},
      {0x62, "Pascal"},
      {0x70, "Volta"},
      {0x72, "Xavier"},
      {0x75, "Turing"},
      {0x80, "Ampere"},
      {0x86, "Ampere"},
      {0x87, "Ampere"},
      {0x90, "Hopper"},
      {-1, "Graphics Device"}};

  int index = 0;

  while (nGpuArchNameSM[index].SM != -1) {
    if (nGpuArchNameSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchNameSM[index].name;
    }

    index++;
  }

  // If we don't find the values, we default use the previous one
  // to run properly
  printf(
      "MapSMtoArchName for SM %d.%d is undefined."
      "  Default to use %s\n",
      major, minor, nGpuArchNameSM[index - 1].name);
  return nGpuArchNameSM[index - 1].name;
}


void CudaHelper::__getLastCudaError(const char *errorMessage, const char *file, const int line) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr,
            "%s(%i) : getLastCudaError() CUDA error :"
            " %s : (%d) %s.\n",
            file, line, errorMessage, static_cast<int>(err),
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// Initialization code to find the best CUDA Device
int CudaHelper::findCudaDevice() {
  int devID = 0;
	// pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	checkCudaErrors(hipSetDevice(devID));
	int major = 0, minor = 0;
	checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
	checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
	printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
			devID, _ConvertSMVer2ArchName(major, minor), major, minor);


  return devID;
}