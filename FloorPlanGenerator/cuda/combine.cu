#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>
#include <stdint.h>
#include <algorithm>
#include <filesystem>
#include <fstream>

#include "helper.cuh"
#include "combine.cuh"
#include "common.cuh"
#include "process.h"
#include "../lib/cvHelper.h"
#include "../lib/globals.h"



// Sorry, had to do it this way to make the reduce the cuda kernel registers usage
// #define check_overlap(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right) ((a_up >= b_up && a_up < b_down && a_left < b_right && a_left >= b_left) || (a_up >= b_up && a_up < b_down && a_right >= b_right && a_left <= b_left) || (a_up >= b_up && a_up < b_down && a_right <= b_right && a_right > b_left) || (a_down >= b_down && a_left < b_right && a_left >= b_left && a_up <= b_up) || (a_down >= b_down && a_up <= b_up && a_right <= b_right && a_right > b_left) || (a_left < b_right && a_left >= b_left && a_down > b_up && a_down <= b_down) || (a_right >= b_right && a_down > b_up && a_down <= b_down && a_left <= b_left) || (b_right >= a_right && b_up >= a_up && b_up < a_down && b_left <= a_left) || (b_right >= a_right && b_down > a_up && b_down <= a_down && b_left <= a_left) || (b_up >= a_up && b_up < a_down && b_left >= a_left && b_left < a_right) || (b_up >= a_up && b_up < a_down && b_right > a_left && b_right <= a_right) || (b_down >= a_down && b_left >= a_left && b_left < a_right && b_up <= a_up) || (b_down >= a_down && b_right > a_left && b_right <= a_right && b_up <= a_up) || (b_left >= a_left && b_left < a_right && b_down > a_up && b_down <= a_down) || (a_down > b_up && a_down <= b_down && a_right <= b_right && a_right > b_left) || (b_right > a_left && b_right <= a_right && b_down > a_up && b_down <= a_down))

// __device__
// uint8_t check_overlap(const int a_up, const int a_down, const int a_left, const int a_right, 
// 	const int b_up, const int b_down, const int b_left, const int b_right){
// 	if(((a_down > b_up && a_down <= b_down) ||
// 	(a_up  >= b_up && a_up < b_down)) &&
// 	((a_right > b_left && a_right <= b_right) ||
// 	(a_left  >= b_left && a_left  <  b_right) ||
// 	(a_left  <= b_left && a_right >= b_right))){
// 		return 0;
// 	}

// 	else if(((b_down > a_up && b_down <= a_down) ||
// 	(b_up >= a_up && b_up < a_down)) &&
// 	((b_right > a_left && b_right <= a_right) ||
// 	(b_left  >= a_left && b_left  <  a_right) ||
// 	(b_left  <= a_left && b_right >= a_right))){
// 		return 0;
// 	}

// 	else if(((a_right > b_left && a_right <= b_right) ||
// 	(a_left >= b_left && a_left < b_right)) &&
// 	((a_down > b_up && a_down <= b_down) ||
// 	(a_up  >= b_up && a_up   <  b_down) ||
// 	(a_up  <= b_up && a_down >= b_down))){
// 		return 0;
// 	}

// 	else if(((b_right > a_left && b_right <= a_right) ||
// 	(b_left >= a_left && b_left < a_right)) &&
// 	((b_down > a_up && b_down <= a_down) ||
// 	(b_up  >= a_up && b_up   <  a_down) ||
// 	(b_up  <= a_up && b_down >= a_down))){
// 		return 0;
// 	}

// 	return 1;
// }


__global__ 
void k_createPts(int16_t *d_a, int16_t *d_b, int *d_res, int *d_adj, const int qtd_a, const int qtd_b, const int a_offset) {
	// Block and thread indexes 	
	// Each blockIdx.x iterates over a fixed number (num_a) of A layouts (blockIdx.y), 
	// that iterates over Nconn connections (blockIdx.z). Each threadIdx.x represents
	// a Layout B design inside the blockIdx.x block 

	//K represents the connection (from 0 to 15, skipping 0, 5, 10 and 15)
	const int k = blockIdx.z + 1 + blockIdx.z/4; 
	int a_idx = blockIdx.y + a_offset; //layout A index
	int b_idx = blockIdx.x * blockDim.x + threadIdx.x; //layout B index
	const uint64_t res_idx = ((blockIdx.y * qtd_b * __COMBINE_CONN) + (b_idx * __COMBINE_CONN) + blockIdx.z) * __SIZE_RES;

	// Check bounds
	if(b_idx >= qtd_b || blockIdx.y >= qtd_a){
		return;
	}

	a_idx *= __SIZE_A_DISK;
	b_idx *= __SIZE_B_DISK;

	// Load A into shared memory
	__shared__ int16_t a[__SIZE_A_DISK];
	if(threadIdx.x < __SIZE_A_DISK){
		a[threadIdx.x] = d_a[a_idx + threadIdx.x];
	}
	
	__shared__ int req_adj[__SIZE_ADJ];
	if(threadIdx.x < __SIZE_ADJ){
		req_adj[threadIdx.x] = d_adj[threadIdx.x];
	}

  	__syncthreads();

	// Load B into local memory
	int16_t b[__SIZE_B_DISK];
	for(int i = 0; i < __SIZE_B_DISK; i++){
		b[i] = d_b[b_idx + i];
	}

	// Extract source and destination connections from k
	const int srcConn = k & 0b11;
	const int dstConn = (k >> 2) & 0b11;

	// Get X axis connection points from layout A and B
	int dst = 0;
	int src = 0;
	if(dstConn == 0 || dstConn == 2)
		dst = b[0];
	else 
		dst = b[2];

	if(srcConn == 0 || srcConn == 2)
		src = a[__SIZE_A_LAYOUT - 4];
	else 
		src = a[__SIZE_A_LAYOUT - 2];


	//Move layout B in the X axis by diffX points
	const int diffX = src - dst;
	for(int i = 0; i < __SIZE_B_LAYOUT; i+=2){
		b[i] += diffX;
	}

	// Get Y axis connection points from layout A and B
	if(dstConn == 0 || dstConn == 1)
		dst = b[1];
	else 
		dst = b[3];
		
	if(srcConn == 0 || srcConn == 1)
		src = a[__SIZE_A_LAYOUT - 3];
	else 
		src = a[__SIZE_A_LAYOUT - 1];

	//Move layout B in the Y axis by diffY points
	const int diffY = src - dst;
	for(int i = 1; i < __SIZE_B_LAYOUT; i+=2){
		b[i] += diffY;
	}

	// Find the bounding box of B
	int minH = 5000, maxH = -5000;
	int minW = 5000, maxW = -5000;
	for(int i = 0; i < __SIZE_B_LAYOUT; i+=4){
		if(b[i + __UP] < minH)
			minH = b[i + __UP];
		if(b[i + __DOWN] > maxH)
			maxH = b[i + __DOWN];
		if(b[i] < minW)
			minW = b[i];
		if(b[i + __RIGHT] > maxW)
			maxW = b[i + __RIGHT];
	}

	//left, up, right, down
	// Find the bounding box of A and check overlaping
	int connections[__COMBINE_N_A + __COMBINE_N_B];
	for(int i = 0; i < __COMBINE_N_A  + __COMBINE_N_B; i++){
		connections[i] = 1 << i;
	}

	for(int i = 0; i < __SIZE_A_LAYOUT; i+=4){
		const int a_left = a[i];
		const int a_up = a[i + __UP];
		const int a_down = a[i + __DOWN];
		const int a_right = a[i + __RIGHT];

		if(a_up < minH)
			minH = a_up;
		if(a_down > maxH)
			maxH = a_down;
		if(a_left < minW)
			minW = a_left;
		if(a_right > maxW)
			maxW = a_right;

		for(int j = 0; j < __SIZE_B_LAYOUT; j+=4){
			const int b_left = b[j];
			const int b_up = b[j + __UP];
			const int b_down = b[j + __DOWN];
			const int b_right = b[j + __RIGHT];

			// if(!check_overlap(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right))
			// 	return;
			
			if(check_adjacency(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right)){
				connections[i/4] |= 1 << (j/4) + __COMBINE_N_A;
				connections[(j/4) + __COMBINE_N_A] |= 1 << (i/4); 
			}
		}
	}

	for(int i = 0; i < __SIZE_A_LAYOUT; i+=4){
		const int a_left = a[i];
		const int a_up = a[i + __UP];
		const int a_down = a[i + __DOWN];
		const int a_right = a[i + __RIGHT];

		for(int j = 0; j < __SIZE_A_LAYOUT; j+=4){
			const int b_left = a[j];
			const int b_up = a[j + __UP];
			const int b_down = a[j + __DOWN];
			const int b_right = a[j + __RIGHT];

			if(check_adjacency(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right)){
				connections[i/4] |= 1 << (j/4);
				connections[j/4] |= 1 << (i/4); 
			}
		}
	}

	for(int i = 0; i < __SIZE_B_LAYOUT; i+=4){
		const int a_left = b[i];
		const int a_up = b[i + __UP];
		const int a_down = b[i + __DOWN];
		const int a_right = b[i + __RIGHT];

		for(int j = 0; j < __SIZE_B_LAYOUT; j+=4){
			const int b_left = b[j];
			const int b_up = b[j + __UP];
			const int b_down = b[j + __DOWN];
			const int b_right = b[j + __RIGHT];

			if(check_adjacency(a_up, a_down, a_left, a_right, b_up, b_down, b_left, b_right)){
				connections[(i/4) + __COMBINE_N_A] |= 1 << ((j/4) + __COMBINE_N_A);
				connections[(j/4) + __COMBINE_N_A] |= 1 << ((i/4) + __COMBINE_N_A); 
			}
		}
	}

	const int a_rid_idx = a[__SIZE_A_LAYOUT];
	const int b_rid_idx = b[__SIZE_B_LAYOUT];

	int adj[__SIZE_ADJ_TYPES]; //Rid connections from the specific rId
	int adj_count[__SIZE_ADJ_TYPES]; //Idx of each room from the specific rId
	for(int i = 0; i < __SIZE_ADJ_TYPES; i++){
		adj[i] = 0;
		adj_count[i] = 0;
	}

	for(int i = 0; i < __COMBINE_N_A; i++){
		const int rplannyId = (a_rid_idx >> (i * __RID_BITS_SIZE)) & __RID_BITS;
		adj_count[rplannyId] |= 1 << i;
		adj[rplannyId] |= connections[i];
	}
	
	for(int i = 0; i < __COMBINE_N_B; i++){
		const int rplannyId = (b_rid_idx >> (i * __RID_BITS_SIZE)) & __RID_BITS;
		adj_count[rplannyId] |= 1 << (i + __COMBINE_N_A);
		adj[rplannyId] |= connections[i + __COMBINE_N_A];
	}

	for(int i = 0; i < __SIZE_ADJ_TYPES; i++){
		for(int j = 0; j < __SIZE_ADJ_TYPES; j++){
			const int req_adj_idx = i*__SIZE_ADJ_TYPES + j;
			// if(req_adj[req_adj_idx] == REQ_ANY && !(adj[j] & adj_count[i]))
			// 	return;

			// if(req_adj[req_adj_idx] == REQ_ALL && (adj[j] & adj_count[i]) != adj_count[i])
			// 	return;
		}
	}

	for(int i = 0; i < __COMBINE_N_A + __COMBINE_N_B; i++){
		const int conns = connections[i];
		for(int j = i + 1; j < __COMBINE_N_A + __COMBINE_N_B; j++){
			if(connections[j] & 1 << i)
				connections[j] |= conns;
		}
	}

	// if(connections[__CONN_CHECK_IDX] != __CONN_CHECK)
	// 	return;

	// if(res_idx == 1660128){
	// 	printf("\n\nbx: %d, by: %d, bz: %d, tx: %d, ty: %d, tz: %d\n\n",
	// 			blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);

	// 	printf("a: (%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n\n", 
	// 	a[0], a[1], a[2], a[3], 
	// 	a[4], a[5], a[6], a[7], 
	// 	a[8], a[9], a[10], a[11]);

	// 	printf("b: (%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n(%d, %d), (%d, %d)\n\n\n", 
	// 	b[0], b[1], b[2], b[3], 
	// 	b[4], b[5], b[6], b[7], 
	// 	b[8], b[9], b[10], b[11]);
	// }

	// d_res[res_idx] = a[0];
	// d_res[res_idx + 1] = a[1];
	// d_res[res_idx + 2] = a[2];
	// d_res[res_idx + 3] = a[3];
	// d_res[res_idx + 4] = a[4];
	// d_res[res_idx + 5] = a[5];
	// d_res[res_idx + 6] = a[6];
	// d_res[res_idx + 7] = a[7];
	// d_res[res_idx + 8] = a[8];
	// d_res[res_idx + 9] = a[9];
	// d_res[res_idx + 10] = a[10];
	// d_res[res_idx + 11] = a[11];

	// d_res[res_idx + 12] = b[0];
	// d_res[res_idx + 13] = b[1];
	// d_res[res_idx + 14] = b[2];
	// d_res[res_idx + 15] = b[3];
	// d_res[res_idx + 16] = b[4];
	// d_res[res_idx + 17] = b[5];
	// d_res[res_idx + 18] = b[6];
	// d_res[res_idx + 19] = b[7];
	// d_res[res_idx + 20] = b[8];
	// d_res[res_idx + 21] = b[9];
	// d_res[res_idx + 22] = b[10];
	// d_res[res_idx + 23] = b[11];
	d_res[res_idx] = maxH - minH;
	d_res[res_idx + 1] = maxW - minW;
	d_res[res_idx + 2] = a_idx;
	d_res[res_idx + 3] = b_idx;
}


int* CudaCombine::createDeviceAdjArray(const std::vector<int>& allReqAdj){
	int* d_adj;
	const size_t mem_size = __SIZE_ADJ * sizeof(int);
	checkCudaErrors(hipMalloc((void **)&d_adj, mem_size));

	int* h_adj = (int*)(allReqAdj.data());
	checkCudaErrors(hipMemcpy(d_adj, h_adj, mem_size, hipMemcpyHostToDevice));

	std::cout << "mem size adj: " << mem_size << ", (MB): " << ((float)mem_size)/1024.0/1024.0 << std::endl;
	return d_adj;
}


int16_t* CudaCombine::createDeviceCoreLayoutsArray(const std::vector<int16_t>& pts){
	int16_t* d_pts;
	const size_t mem_size = pts.size() * sizeof(int16_t);
	checkCudaErrors(hipMalloc((void **)&d_pts, mem_size));

	int16_t* h_pts = (int16_t*)(pts.data());
	checkCudaErrors(hipMemcpy(d_pts, h_pts, mem_size, hipMemcpyHostToDevice));

	std::cout << "mem size core layout: " << mem_size << ", (MB): " << ((float)mem_size)/1024.0/1024.0 << std::endl;
	return d_pts;
}

int* CudaCombine::createDeviceResArray(const size_t mem_size) {
	int *d_res = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_res, mem_size));
	checkCudaErrors(hipMemset(d_res, -1, mem_size));

	std::cout << "mem size res: " << mem_size << ", (MB): " << ((float)mem_size)/1024.0/1024.0 << std::endl;
	return d_res;
}

void CudaCombine::freeDeviceArrays(int* adj, int* res, int16_t* a, int16_t* b) {
	checkCudaErrors(hipFree(a));
	checkCudaErrors(hipFree(b));
	checkCudaErrors(hipFree(adj));
	checkCudaErrors(hipFree(res));
}

void CudaCombine::createPts(
		const size_t res_mem_size,
		const long NConn,
		const long num_a,
		const long qtd_b,
		const long a_offset,
		const long num_blocks,
		const long num_threads,
		int* h_res,
		int* d_adj,
		int* d_res,
		int16_t* d_a,
		int16_t* d_b) 
	{

	dim3 grid(num_blocks, num_a, NConn);
	dim3 threads(num_threads, 1, 1);

	checkCudaErrors(hipMemset(d_res, -1, res_mem_size));

	k_createPts<<<grid, threads>>>(d_a, d_b, d_res, d_adj, num_a, qtd_b, a_offset);
	hipDeviceSynchronize();

	checkCudaErrors(hipMemcpy(h_res, d_res, res_mem_size, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
}